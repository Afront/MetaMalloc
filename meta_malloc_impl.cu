#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>

// https://stackoverflow.com/a/28166605
#if defined(__GNUC__) || defined(__GNUG__)
	#define ALL_DEVICES __attribute__ ((device)) __attribute__ ((host))
	#define DEVICE __attribute__ ((device))
	#define HOST __attribute__ ((host))
	#define GLOBAL __attribute__ ((global))
#elif defined(_MSC_VER)
// https://learn.microsoft.com/en-us/previous-versions/visualstudio/visual-studio-2012/dabb5z75(v=vs.110)?redirectedfrom=MSDN
// https://stackoverflow.com/questions/28411283/dealing-with-attribute-in-msvc
/*	#define ALL_DEVICES __declspec(device) __declspec(host)
	#define DEVICE __declspec(device)
	#define HOST __declspec(host)
	#define GLOBAL __declspec(global)*/
	#ifdef __HIPCC__
		#define ALL_DEVICES __device__ __host__
		#define DEVICE __device__
		#define HOST __host__
		#define GLOBAL __global__
	#else
		#define ALL_DEVICES
		#define DEVICE
		#define HOST
		#define GLOBAL
	#endif
#endif

struct LogDataArray {
	char* kernel_name;
	const dim3 block_dim;
	const dim3 grid_dim;

	int64_t* clock_arr;
	int32_t* thread_id_arr;
	int32_t* block_id_arr;
	void** address_arr;
	size_t* memory_size_arr;
	const char** type_arr;

	ALL_DEVICES size_t length() {
		return block_dim.x * block_dim.y * block_dim.z *
			grid_dim.x * grid_dim.y * grid_dim.z;
	}

	HOST LogDataArray(std::string kernel_name_str, const dim3& grid_dim, const dim3& block_dim) : block_dim(block_dim), grid_dim(grid_dim) {
		CHECK_ERROR(hipMallocManaged(&kernel_name, sizeof(char) * kernel_name_str.size()));

		size_t i = 0;
		for(auto&& c : kernel_name_str) {
			kernel_name[i] = c;
			i++;
		}

		CHECK_ERROR(hipMallocManaged(&clock_arr, sizeof(int64_t) * length()));
		CHECK_ERROR(hipMallocManaged(&thread_id_arr, sizeof(int32_t) * length()));
		CHECK_ERROR(hipMallocManaged(&block_id_arr, sizeof(int32_t) * length()));
		CHECK_ERROR(hipMallocManaged(&address_arr, sizeof(void*) * length()));
		CHECK_ERROR(hipMallocManaged(&memory_size_arr, sizeof(size_t) * length()));
		CHECK_ERROR(hipMallocManaged(&type_arr, sizeof(char*) * length()));
	}

	HOST void free() {
		CHECK_ERROR(hipFree(kernel_name));
		CHECK_ERROR(hipFree(clock_arr));
		CHECK_ERROR(hipFree(thread_id_arr));
		CHECK_ERROR(hipFree(block_id_arr));
		CHECK_ERROR(hipFree(address_arr));		
		CHECK_ERROR(hipFree(memory_size_arr));
		CHECK_ERROR(hipFree(type_arr));
	
	}

	ALL_DEVICES void print_at_index(size_t i) {
		printf(
			"kernel name: %s,"
			"clock: %li,"
			"threadIdx.x: %d,"
			"blockIdx.x: %d,"
			"blockDim.x: %d,"
			"blockDim.y: %d,"
			"blockDim.z: %d,"
			"address: %p,\n",
			"memory_size: %d\n",

			kernel_name,
			clock_arr[i],
			thread_id_arr[i],
			block_id_arr[i],
			block_dim.x,
			block_dim.y,
			block_dim.z,
			address_arr[i],
			memory_size_arr[i]
		);
	}

	HOST std::string data_to_s(size_t i) {
		// CSV format
		// Kernel name, grid dim, block dim, type, clock, thread idx, block idx, address, memory size

		std::stringstream string_stream;

		string_stream << 
			kernel_name  << ',' <<
			grid_dim.x << ',' <<
			grid_dim.y << ',' <<
			grid_dim.z << ',' <<
			block_dim.x << ',' <<
			block_dim.y << ',' <<
			block_dim.z << ',' <<

			type_arr[i] << ',' <<

			clock_arr[i] << ',' <<
			thread_id_arr[i] << ',' <<
			block_id_arr[i] << ',' <<
			address_arr[i] // << ',' <<
			// memory_size_arr[i]
			<< std::endl;

		return string_stream.str();
	}



	HOST void write_to_file(std::string filename) {
		for (size_t i = 0; i < length(); i++){
			std::cout << data_to_s(i) << '\n'; // -> write to file
		}


	}
};

class MemoryManager {
	MemoryAllocator memory_allocator;

public:	
	HOST MemoryManager(size_t size) : memory_allocator(MemoryAllocator(size)) {}

	DEVICE __forceinline__ void* malloc(
		size_t size,
		LogDataArray log_data
	) {
		// 3 "heavy" calls: malloc, clock64 read, printf
		// not sure how to order

		// technically should benchmark here instead
		auto pointer = memory_allocator.malloc(size);
		// should end benchmark here

		// printf("pointer %p\n", pointer);

		auto tid = threadIdx.x + blockIdx.x * blockDim.x;

		log_data.clock_arr[tid] = clock64();
		log_data.thread_id_arr[tid] = threadIdx.x;
		log_data.block_id_arr[tid] = blockIdx.x;
		log_data.address_arr[tid] = pointer;
		log_data.memory_size_arr[tid] = size;
		log_data.type_arr[tid] = "malloc";


		log_data.print_at_index(tid);


		return pointer;
	}

	DEVICE __forceinline__ void free(void* pointer) {
		return memory_allocator.free(pointer);
	}
};
