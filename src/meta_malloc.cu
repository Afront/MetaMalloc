#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <tl/expected.hpp>

// https://stackoverflow.com/a/28166605
#if defined(__GNUC__) || defined(__GNUG__)
	#define ALL_DEVICES __attribute__ ((device)) __attribute__ ((host))
	#define DEVICE __attribute__ ((device))
	#define HOST __attribute__ ((host))
	#define GLOBAL __attribute__ ((global))
#elif defined(_MSC_VER)
// https://learn.microsoft.com/en-us/previous-versions/visualstudio/visual-studio-2012/dabb5z75(v=vs.110)?redirectedfrom=MSDN
// https://stackoverflow.com/questions/28411283/dealing-with-attribute-in-msvc
/*	#define ALL_DEVICES __declspec(device) __declspec(host)
	#define DEVICE __declspec(device)
	#define HOST __declspec(host)
	#define GLOBAL __declspec(global)*/
	#ifdef __HIPCC__
		#define ALL_DEVICES __device__ __host__
		#define DEVICE __device__
		#define HOST __host__
		#define GLOBAL __global__
	#else
		#define ALL_DEVICES
		#define DEVICE
		#define HOST
		#define GLOBAL
	#endif
#endif

using CUDAResult = tl::expected<bool, hipError_t>;
/*
CUDAResult to_CUDAResult(hipError_t error) {

	return (error == hipSuccess) ? true : tl::make_unexpected(error);
}
*/

HOST void error_check(hipError_t error) {
	if (error != hipSuccess){
		std::cout << "Error: " << hipGetErrorString(error) << '\n';
		exit(1);
	}
}

struct LogDataArray {
	char* kernel_name;
	const dim3 block_dim;
	const dim3 grid_dim;

	int64_t* clock_arr;
	int32_t* thread_id_arr;
	int32_t* block_id_arr;
	void** address_arr;
	size_t* memory_size_arr;
	const char** type_arr;

	ALL_DEVICES size_t length() {
		return block_dim.x * block_dim.y * block_dim.z *
			grid_dim.x * grid_dim.y * grid_dim.z;
	}

	HOST LogDataArray(std::string kernel_name_str, const dim3& grid_dim, const dim3& block_dim) : block_dim(block_dim), grid_dim(grid_dim) {
		// to_CUDAResult(hipMallocManaged(&kernel_name, sizeof(char) * kernel_name_str.size()));
		error_check(hipMallocManaged(&kernel_name, sizeof(char) * kernel_name_str.size()));

		size_t i = 0;
		for(auto&& c : kernel_name_str) {
			kernel_name[i] = c;
			i++;
		}

		error_check(hipMallocManaged(&clock_arr, sizeof(int64_t) * length()));
		error_check(hipMallocManaged(&thread_id_arr, sizeof(int32_t) * length()));
		error_check(hipMallocManaged(&block_id_arr, sizeof(int32_t) * length()));
		error_check(hipMallocManaged(&address_arr, sizeof(void*) * length()));
		error_check(hipMallocManaged(&memory_size_arr, sizeof(size_t) * length()));
		error_check(hipMallocManaged(&type_arr, sizeof(char*) * length()));
	}

	HOST void free() {
		error_check(hipFree(kernel_name));
		error_check(hipFree(clock_arr));
		error_check(hipFree(thread_id_arr));
		error_check(hipFree(block_id_arr));
		error_check(hipFree(address_arr));		
		error_check(hipFree(memory_size_arr));
		error_check(hipFree(type_arr));
	
	}

	ALL_DEVICES void print_at_index(size_t i) {
		printf(
			"kernel name: %s,"
			"clock: %li,"
			"threadIdx.x: %d,"
			"blockIdx.x: %d,"
			"blockDim.x: %d,"
			"blockDim.y: %d,"
			"blockDim.z: %d,"
			"address: %p"
			"memory_size: %lu"
//			""
			"\n",
			kernel_name,
			clock_arr[i],
			thread_id_arr[i],
			block_id_arr[i],
			block_dim.x,
			block_dim.y,
			block_dim.z,
			address_arr[i],
			memory_size_arr[i]
		);
	}

	HOST std::string data_to_s(size_t i) {
		// CSV format
		// Kernel name, grid dim, block dim, type, clock, thread idx, block idx, address, memory size

		std::stringstream string_stream;

		string_stream << 
			kernel_name  << ',' <<
			grid_dim.x << ',' <<
			grid_dim.y << ',' <<
			grid_dim.z << ',' <<
			block_dim.x << ',' <<
			block_dim.y << ',' <<
			block_dim.z << ',' <<

			type_arr[i] << ',' <<

			clock_arr[i] << ',' <<
			thread_id_arr[i] << ',' <<
			block_id_arr[i] << ',' <<
			address_arr[i] // << ',' <<
			// memory_size_arr[i]
			<< std::endl;

		return string_stream.str();
	}



	HOST void write_to_file(std::string filename) {
		for (size_t i = 0; i < length(); i++){
			std::cout << data_to_s(i) << '\n'; // -> write to file
		}


	}
};


template <typename MemoryAllocator>
class MemoryManager {
	MemoryAllocator memory_allocator;

public:	
	HOST MemoryManager(size_t size) : memory_allocator(MemoryAllocator(size)) {}

	DEVICE __forceinline__ void* malloc(
		size_t size,
		LogDataArray log_data
	) {
		// 3 "heavy" calls: malloc, clock64 read, printf
		// not sure how to order

		// technically should benchmark here instead
		auto pointer = memory_allocator.malloc(size);
		// should end benchmark here

		// printf("pointer %p\n", pointer);

		auto tid = threadIdx.x + blockIdx.x * blockDim.x;

		log_data.clock_arr[tid] = clock64();
		log_data.thread_id_arr[tid] = threadIdx.x;
		log_data.block_id_arr[tid] = blockIdx.x;
		log_data.address_arr[tid] = pointer;
		log_data.memory_size_arr[tid] = size;
		log_data.type_arr[tid] = "malloc";


		log_data.print_at_index(tid);


		return pointer;
	}

	DEVICE __forceinline__ void free(void* pointer) {
		return memory_allocator.free(pointer);
	}
};
